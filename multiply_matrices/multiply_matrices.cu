
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
//#include <inttypes.h>
#include <stdint.h>

// Structure definitions
typedef struct matrix_ {
    uint16_t     rows;
    uint16_t     columns;
    uint16_t**   data; // data 2D-array will be [row][column]
} matrix_t;

int validate_matrices(matrix_t* a, matrix_t* b) {
    if (a->columns == b->rows) {
        return 0;
    }
    return -1;
}

int set_result_matrix_dimensions(matrix_t* a, matrix_t* b, matrix_t* result) {
    if (validate_matrices(a, b) == 0) {
        result->rows = a->rows;
        result->columns = b->columns;
        return 0;
    }
    return -1;
}

int allocate_matrix_memory(matrix_t* matrix) {
    // FIXME: malloc check? i don't feel like it....
    matrix->data = (uint16_t**) malloc(sizeof(uint16_t*) * matrix->rows);
    for (uint16_t i=0; i<matrix->rows; i++) {
        matrix->data[i] = (uint16_t*) malloc(sizeof(uint16_t) * matrix->columns);
    }
    return 0;
}

int generate_matrix(matrix_t* matrix) {
    allocate_matrix_memory(matrix);
    for (uint16_t i=0; i<matrix->rows; i++) {
        for (uint16_t j=0; j<matrix->columns; j++) {
            matrix->data[i][j] = i+j;   // I just picked some arbitrary number
        }
    }
    return 0;
}

int print_matrix(matrix_t* matrix) {
    for (uint16_t i=0; i<matrix->rows; i++) {
        printf("[ ");
        for (uint16_t j=0; j<matrix->columns; j++) {
            printf("%d ", matrix->data[i][j]);
        }
        printf("]\n");
    }

    return 0;
}

int main() {
    /*
    TODO: perform matrix multplication using
        - single thread
        - multiprocessing
        - multithreading
        - GPU (figure out techniques?)
    */
    matrix_t a, b;

    a.rows = 10;
    a.columns = 10;

    b.rows = 10;
    b.columns = 10;

    generate_matrix(&a);
    generate_matrix(&b);

    print_matrix(&a);

    return 0;
}
