
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
//#include <inttypes.h>
#include <stdint.h>
#include <time.h>

// Structure definitions
typedef struct matrix_ {
    uint16_t     rows;
    uint16_t     columns;
    uint16_t**   data; // data 2D-array will be [row][column]
} matrix_t;

int validate_matrices(matrix_t* a, matrix_t* b) {
    if (a->columns == b->rows) {
        return 0;
    }
    return -1;
}

int set_result_matrix_dimensions(matrix_t* a, matrix_t* b, matrix_t* result) {
    if (validate_matrices(a, b) == 0) {
        result->rows = a->rows;
        result->columns = b->columns;
        return 0;
    }
    return -1;
}

int allocate_matrix_memory(matrix_t* matrix) {
    // FIXME: malloc check? i don't feel like it....
    matrix->data = (uint16_t**) malloc(sizeof(uint16_t*) * matrix->rows);
    for (uint16_t i=0; i<matrix->rows; i++) {
        matrix->data[i] = (uint16_t*) malloc(sizeof(uint16_t) * matrix->columns);
    }
    return 0;
}

int generate_matrix(matrix_t* matrix) {
    allocate_matrix_memory(matrix);
    for (uint16_t i=0; i<matrix->rows; i++) {
        for (uint16_t j=0; j<matrix->columns; j++) {
            matrix->data[i][j] = i+j;   // I just picked some arbitrary number
        }
    }
    return 0;
}

int generate_result_matrix(matrix_t* a, matrix_t* b, matrix_t* result) {
    set_result_matrix_dimensions(a, b, result);
    allocate_matrix_memory(result);
    return 0;
}

int print_matrix(matrix_t* matrix) {
    for (uint16_t i=0; i<matrix->rows; i++) {
        printf("[ ");
        for (uint16_t j=0; j<matrix->columns; j++) {
            printf("%d ", matrix->data[i][j]);
        }
        printf("]\n");
    }

    return 0;
}

uint16_t calculate_result_element(uint16_t row, uint16_t column, matrix_t* a, matrix_t* b) {
    uint16_t max = a->columns;
    uint16_t sum = 0;
    for (uint16_t count=0; count<max; count++) {
        sum += a->data[row][count] * b->data[count][column];
    }
    return sum;
}

int single_thread_matrix_multiply(matrix_t* a, matrix_t* b, matrix_t* result) {
    for (uint16_t i=0; i<result->rows; i++) {
        for (uint16_t j=0; j<result->columns; j++) {
            result->data[i][j] = calculate_result_element(i, j, a, b);
        }
    }
    return 0;
}

int main() {
    /*
    TODO: perform matrix multplication using
        - single thread
        - multiprocessing
        - multithreading
        - GPU (figure out techniques?)
    */
    matrix_t a, b, result;
    time_t start, end;

    a.rows = 100;
    a.columns = 100;

    b.rows = 100;
    b.columns = 100;

    generate_matrix(&a);
    generate_matrix(&b);

    generate_result_matrix(&a, &b, &result);
    
    // print_matrix(&a);
    // printf("\n");
    // print_matrix(&b);
    // printf("\n");

    printf("single_thread_matrix_multiply()\n");
    time(&start);
    single_thread_matrix_multiply(&a, &b, &result);
    time(&end);
    // print_matrix(&result);

    // FIXME: it no work...
    printf("Time to perform: %f seconds\n", difftime(end, start));

    return 0;
}
