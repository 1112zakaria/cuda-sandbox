#include <stdlib.h>
#include <stdio.h>

void print_num_gpus() {
    int count;
    if (hipGetDeviceCount(&count) == 0) {
        printf("Num GPUs: %d\n", count);
    }
}

void print_current_device() {
    int device;

    if (hipGetDevice(&device) == 0) {
        printf("Current Device: %d\n", device);
    }
}

void print_device_properties(int device_index) {
    // FIXME: use macro for printing device prop w/ \t and \n
    hipDeviceProp_t device_properties;
    if (hipGetDeviceProperties(&device_properties, device_index) == 0) {
        printf("\tDevice ID %d Info:\n", device_index);
        printf("\tName: %s\n", device_properties.name);
        printf("\tTotal Global Memory: %ld bytes\n", device_properties.totalGlobalMem);
        printf("\tClock Rate: %d kHz\n", device_properties.clockRate);
        printf("\tMultiprocessor Count: %d\n", device_properties.multiProcessorCount);
        printf("\tL2 Cache Size: %d bytes\n", device_properties.l2CacheSize);
        printf("\n");
    }
}

void print_all_device_properties() {
    int device_count;

    if (hipGetDeviceCount(&device_count) == 0) {
        printf("Device Properties:\n");
        for (int i=0; i<device_count; i++) {
            print_device_properties(i);
            printf("\n");
        }
    }
}

int main() {

    printf("--DISPLAY GPU INFO--\n");
    print_num_gpus();
    print_current_device();
    print_all_device_properties();
    
    return 0;

}